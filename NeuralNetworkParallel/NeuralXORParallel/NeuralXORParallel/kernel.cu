#include "hip/hip_runtime.h"
#include ""
#include "XORNetwork.h"

#include <hip/device_functions.h>
#include <iostream>
#include <stdio.h>

int main()
{
	XORNetwork network("XORData.json");
	network.trainByBackpropagation(100000, 0.5);

	int o1, o2, input;
	do{
		do{
			printf("\nOperand 1: ");
			std::cin >> input;
		}while(input != 1 && input != 0 && input != -1);
		o1 = input;

		do{
			printf("\nOperand 2: ");
			std::cin >> input;
		}while(input != 1 && input != 0 && input != -1);
		o2 = input;
		
		double result = network.xor(o1, o2);

		printf("\n%d XOR %d = %f\n", o1, o2, result);
	}while(input != -1);
	system("pause");

	hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
