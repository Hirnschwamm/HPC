#include "XORNetwork.h"

#include <cstdio>
#include <tuple>

#include "rapidjson\document.h"
#include "rapidjson\writer.h"
#include "rapidjson\stringbuffer.h"
#include "rapidjson\filereadstream.h"

XORNetwork::XORNetwork(std::string dataPath){
	int inputDim = 2;
	int outputDim = 1;
	
	for(int i = 0; i < inputDim; i++){
		inputLayer.push_back(new Perceptron(0.0f));
	}

	for(int i = 0; i < inputDim; i++){
		hiddenLayer.push_back(new Perceptron(inputLayer));
	}

	for(int i = 0; i < outputDim; i++){
		outputLayer.push_back(new Perceptron(hiddenLayer));
	}

	initjsonTrainingData(dataPath);
}


XORNetwork::~XORNetwork(void)
{
	for(unsigned int i = 0; i < inputLayer.size(); i++){
		delete inputLayer[i];
	}

	for(unsigned int i = 0; i < hiddenLayer.size(); i++){
		delete hiddenLayer[i];
	}

	for(unsigned int i = 0; i < outputLayer.size(); i++){
		delete outputLayer[i];
	}
}

void XORNetwork::trainByBackpropagation(unsigned int passes, double learningRate){
	double gatheredInputData[8];
	double gatheredOutputData[4];
	std::vector<double>* inputData;
	std::vector<double>* outputData;
	int index = 0;
	for(unsigned int i = 0; i < trainingData.size(); i++){
		inputData = &std::get<0>(trainingData[i]);
		outputData = &std::get<1>(trainingData[i]);

		gatheredInputData[index] = inputData->at(0);
		gatheredInputData[index + 1] = inputData->at(1);
		index += 2;

		gatheredOutputData[i] = outputData->at(0);	
	}

	double gatheredWeights[2][4]; //[layer][weight]

	for(unsigned int i = 0; i < hiddenLayer.size(); i++){
		gatheredWeights[0][i * 2] = hiddenLayer[i]->getWeight(0);
		gatheredWeights[0][i * 2 + 1] = hiddenLayer[i]->getWeight(1);
	}
	for(unsigned int i = 0; i < outputLayer.size(); i++){
		gatheredWeights[1][i * 2] = outputLayer[i]->getWeight(0);
		gatheredWeights[1][i * 2 + 1] = outputLayer[i]->getWeight(1);
	}

	double gatheredBiasWeights[2][2];
	for(unsigned int i = 0; i < hiddenLayer.size(); i++){
		gatheredBiasWeights[0][i] = hiddenLayer[i]->getBiasWeight();
	}
	for(unsigned int i = 0; i < outputLayer.size(); i++){
		gatheredBiasWeights[1][i] = outputLayer[i]->getBiasWeight();
	}

	double *dev_input = 0;
    double *dev_output = 0;
    double *dev_weights = 0;
	double *dev_bias = 0;
	double *dev_error = 0;
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&dev_input, 8 * sizeof(double));
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

	cudaStatus = hipMalloc((void**)&dev_output, 4 * sizeof(double));
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

	cudaStatus = hipMalloc((void**)&dev_weights, 8 * sizeof(double));
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

	cudaStatus = hipMalloc((void**)&dev_bias, 4 * sizeof(double));
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

	cudaStatus = hipMalloc((void**)&dev_error, 4 * sizeof(double));
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

	cudaStatus = hipMemcpy(dev_input, gatheredInputData, 8 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

	cudaStatus = hipMemcpy(dev_output, gatheredOutputData, 4 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

	double gatheredWeightsFlattened[8];
	int k = 0;
	for(int i = 0; i < 2; i++){
		for(int j = 0; j < 4; j++){
			gatheredWeightsFlattened[k] = gatheredWeights[i][j];
			k++;
		}
	}
	cudaStatus = hipMemcpy(dev_weights, gatheredWeightsFlattened, 8 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

	double gatheredBiasesFlattened[4];
	k = 0;
	for(int i = 0; i < 2; i++){
		for(int j = 0; j < 2; j++){
			gatheredBiasesFlattened[k] = gatheredBiasWeights[i][j];
			k++;
		}
	}
	cudaStatus = hipMemcpy(dev_bias, gatheredBiasesFlattened, 4 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

	for(unsigned int pass = 0; pass < passes; pass++){
		printf("Training AI... %d. pass\n", pass + 1);

		backpropagationPass<<<1, 4>>>(dev_input, dev_output, dev_weights, 4, dev_bias, 2, dev_error, learningRate);
	}

	double errors[4];
	cudaStatus = hipMemcpy(errors, dev_error, 4 * sizeof(double), hipMemcpyDeviceToHost);
	printf(" Error: 1. Set: %f | 2. Set: %f | 3. Set: %f | 4. Set: %f\n", errors[0], errors[1], errors[2], errors[3]);
	cudaStatus = hipMemcpy(gatheredWeightsFlattened, dev_weights, 8 * sizeof(double), hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpy(gatheredBiasesFlattened, dev_bias, 4 * sizeof(double), hipMemcpyDeviceToHost);

	k = 0;
	for(int i = 0; i < 2; i++){
		for(int j = 0; j < 4; j++){
			gatheredWeights[i][j] = gatheredWeightsFlattened[k];
			k++;
		}
	}

	k = 0;
	for(int i = 0; i < 2; i++){
		for(int j = 0; j < 2; j++){
			gatheredBiasWeights[i][j] = gatheredBiasesFlattened[k];
			k++;
		}
	}

	for(unsigned int i = 0; i < hiddenLayer.size(); i++){
		hiddenLayer[i]->setWeight(0, gatheredWeights[0][i * 2]);
		hiddenLayer[i]->setWeight(1, gatheredWeights[0][i * 2 + 1]);
	}

	for(unsigned int i = 0; i < outputLayer.size(); i++){
		outputLayer[i]->setWeight(0, gatheredWeights[1][i * 2]);
		outputLayer[i]->setWeight(1, gatheredWeights[1][i * 2 + 1]);
	}

	for(unsigned int i = 0; i < hiddenLayer.size(); i++){
		hiddenLayer[i]->setBias(gatheredBiasWeights[0][i]);
	}

	for(unsigned int i = 0; i < outputLayer.size(); i++){
		 outputLayer[i]->setBias(gatheredBiasWeights[1][i]);
	}

	hipFree(dev_input);
	hipFree(dev_output);
	hipFree(dev_weights);
	hipFree(dev_bias);
	
	printf("Done training AI!\n");
}

double XORNetwork::xor(int operand1, int operand2){
	std::vector<double> input;
	input.push_back((double)operand1);
	input.push_back((double)operand2);

	setInput(input);

	return outputLayer[0]->getOutput();
}

void XORNetwork::setInput(std::vector<double>& input){
	for(unsigned int i = 0; i < input.size(); i++){
		inputLayer[i]->setDirectInput(input[i]);
	}
}

void XORNetwork::initjsonTrainingData(std::string path){
	FILE* fp;
	fopen_s(&fp, path.c_str(), "rb"); 
	assert(fp);
	char readBuffer[65536];
	rapidjson::FileReadStream is(fp, readBuffer, sizeof(readBuffer));
	rapidjson::Document jsonDoc;
	jsonDoc.ParseStream(is);
	fclose(fp);
	assert(jsonDoc.IsObject());

	rapidjson::Value& moves = jsonDoc["moves"];
	assert(moves.IsArray());
	std::vector<double> input;
	std::vector<double> output;
	for (rapidjson::SizeType i = 0; i < moves.Size(); i++){
		for(rapidjson::SizeType j = 0; j < moves[i]["input"].Size(); j++){
			input.push_back( moves[i]["input"][j].GetDouble());
		}
		for(rapidjson::SizeType j = 0; j < moves[i]["output"].Size(); j++){
			output.push_back( moves[i]["output"][j].GetDouble());
		}
		trainingData.push_back(std::tuple<std::vector<double>, std::vector<double>>(input, output));
		input.clear();
		output.clear();
	}
}
